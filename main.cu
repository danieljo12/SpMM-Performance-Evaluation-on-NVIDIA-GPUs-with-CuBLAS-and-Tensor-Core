#define USE_CUBLAS

#include "./header.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <cstdlib>
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <numeric>
#include <vector>

int main(int argc, char** argv)
{
    if (argc != 4 && argc != 5) {
        printf("Wrong Inputs! Correct input format: ./spmm_test M K N [Sparsity]\n");
        return 0;
    }
    int M_GLOBAL                    = atoi(argv[1]);
    int K_GLOBAL                    = atoi(argv[2]);
    int N_GLOBAL                    = atoi(argv[3]);
    int MATRIX_A_PRUNING_PERCENTAGE = (argc == 5) ? atoi(argv[4]) : 0;

    hipblasStatus_t cublas_status;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Host memory
    half* A_h            = NULL;  // row major
    half* B_h            = NULL;  // col major
    // Device memory
    half* A            = NULL;
    half* B            = NULL;
    //
    A_h            = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    B_h            = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    if (A_h == NULL || B_h == NULL) {
        printf("Error in CPU Malloc!\n");
        exit(-1);
    }
    hipMalloc(reinterpret_cast<void**>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);
    checkLastCudaError(__LINE__);
    if (A == NULL || B == NULL) {
        printf("Error in hipMalloc!\n");
        exit(-1);
    }
    //
    init_host_matrices(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);

    // printf("Preparing dense data for GPU...\n");
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    //#ifdef USE_CUBLAS
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuBlas...\n");
    half* D_cublas = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);

    // Tensor core not enabled
    hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);
    hipDeviceSynchronize();
    int              m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float      alpha     = 1.0;
    const float      beta      = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    // for (int i = 0; i < WARM_UP_ITERATION; i++) {
    //     cublas_status = hipblasGemmEx(handle,
    //                                  HIPBLAS_OP_T,
    //                                  HIPBLAS_OP_N,
    //                                  m,
    //                                  n,
    //                                  k,
    //                                  &alpha,
    //                                  A,
    //                                  HIP_R_16F,
    //                                  k,
    //                                  B,
    //                                  HIP_R_16F,
    //                                  k,
    //                                  &beta,
    //                                  D_cublas,
    //                                  HIP_R_16F,
    //                                  m,
    //                                  HIP_R_32F,
    //                                  CuBlasALG);
    //     checkCublasError(cublas_status, __LINE__);
    // }
    hipEventRecord(start);
    // for (int i = 0; i < BENCHMARK_ITERATION; i++)
    //     hipblasGemmEx(handle,
    //                  HIPBLAS_OP_T,
    //                  HIPBLAS_OP_N,
    //                  m,
    //                  n,
    //                  k,
    //                  &alpha,
    //                  A,
    //                  HIP_R_16F,
    //                  k,
    //                  B,
    //                  HIP_R_16F,
    //                  k,
    //                  &beta,
    //                  D_cublas,
    //                  HIP_R_16F,
    //                  m,
    //                  HIP_R_32F,
    //                  CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas = 0;
    hipEventElapsedTime(&milliseconds_cublas, start, stop);
    milliseconds_cublas = milliseconds_cublas / BENCHMARK_ITERATION;
    float tflops_cublas =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas / 1000.))
        / 1e12;
    // Tensor core enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    hipDeviceSynchronize();
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     m,
                                     n,
                                     k,
                                     &alpha,
                                     A,
                                     HIP_R_16F,
                                     k,
                                     B,
                                     HIP_R_16F,
                                     k,
                                     &beta,
                                     D_cublas,
                                     HIP_R_16F,
                                     m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        hipblasGemmEx(handle,
                     HIPBLAS_OP_T,
                     HIPBLAS_OP_N,
                     m,
                     n,
                     k,
                     &alpha,
                     A,
                     HIP_R_16F,
                     k,
                     B,
                     HIP_R_16F,
                     k,
                     &beta,
                     D_cublas,
                     HIP_R_16F,
                     m,
                     HIP_R_32F,
                     CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas_tc = 0;
    hipEventElapsedTime(&milliseconds_cublas_tc, start, stop);
    milliseconds_cublas_tc = milliseconds_cublas_tc / BENCHMARK_ITERATION;
    float tflops_cublas_tc = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                 / (milliseconds_cublas_tc / 1000.))
                             / 1e12;
    half* D_cublas_h = NULL;  // col major
    D_cublas_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_cublas);
    /////////////////////////////////////////////////////////////////////////////////////////////////
//#endif

    printf("******************************************Problem Size******************************************\n");
    printf("M: %d N: %d K: %d Pruning Rate: %d\n",
           M_GLOBAL,
           N_GLOBAL,
           K_GLOBAL,
           MATRIX_A_PRUNING_PERCENTAGE);
// printf("******************************************Performance*******************************************\n");
#ifdef USE_CUBLAS
    // PrintPerformance("CuBlas_SIMT", milliseconds_cublas, tflops_cublas, 0.0, 0.0);
    PrintPerformance("CuBlas_TC", milliseconds_cublas_tc, tflops_cublas_tc, 0.0, 0.0);
#endif

    free(D_cublas_h);
    free(A_h);
    free(B_h);
    hipFree(A);
    hipFree(B);

    return 0;
}
